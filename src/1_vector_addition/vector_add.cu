#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// CPU implementation of vector addition
void vectorAddCPU(const float* A, const float* B, float* C, int n) {
    for (int i = 0; i < n; i++) {
        C[i] = A[i] + B[i];
    }
}

// CUDA kernel for vector addition
__global__ void vectorAddGPU(const float* A, const float* B, float* C, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

// Function to check CUDA errors
void checkCudaError(hipError_t error, const char* msg) {
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s: %s\n", msg, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Vector size
    const int n = 10e7;
    size_t size = n * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);
    float *h_C_CPU = (float*)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < n; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc(&d_A, size), "hipMalloc A");
    checkCudaError(hipMalloc(&d_B, size), "hipMalloc B");
    checkCudaError(hipMalloc(&d_C, size), "hipMalloc C");

    // Copy data from host to device
    checkCudaError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "hipMemcpy A");
    checkCudaError(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "hipMemcpy B");

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start time
    hipEventRecord(start);

    vectorAddGPU<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate GPU execution time
    float gpuTime = 0;
    hipEventElapsedTime(&gpuTime, start, stop);

    // Copy result back to host
    checkCudaError(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost), "hipMemcpy C");

    // CPU implementation timing
    clock_t cpuStart = clock();
    vectorAddCPU(h_A, h_B, h_C_CPU, n);
    clock_t cpuEnd = clock();
    float cpuTime = ((float)(cpuEnd - cpuStart)) / CLOCKS_PER_SEC * 1000.0f;

    // Verify results
    bool correct = true;
    for (int i = 0; i < n; i++) {
        if (fabs(h_C[i] - h_C_CPU[i]) > 1e-5) {
            correct = false;
            break;
        }
    }

    // Print results
    printf("Vector size: %d\n", n);
    printf("GPU Time: %.3f ms\n", gpuTime);
    printf("CPU Time: %.3f ms\n", cpuTime);
    printf("Speedup: %.2fx\n", cpuTime / gpuTime);
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_CPU);

    return 0;
}
